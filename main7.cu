#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

//#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

#ifndef __HIPCC__
#define __HIPCC__
#endif 

#define sqr(x) ((x)*(x))
#define SX 512
#define SY 512


//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)

using namespace std;
using namespace cv;

void set_cufftcomplex(hipfftComplex* cuconp, double* Re, double* Im, int size) {
	for (int i = 0; i < size; i++) {
		cuconp[i] = make_hipComplex((float)Re[i], (float)Im[i]);
	}
}


void fft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
	hipfftDestroy(plan);


}


void ifft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
	hipfftDestroy(plan);
}

void cufftcom_to_mycom(My_ComArray_2D* out, hipfftComplex* in, int s) {
	for (int i = 0; i < s; i++) {
		out->Re[i] = (double)hipCrealf(in[i]);
		out->Im[i] = (double)hipCimagf(in[i]);

	}
}



__global__ void Hcudaf(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
{



	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idy < y && idx < x) {
		Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
		Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
	}


}

__global__ void  shiftf(float* ore, float* oim, float* re, float* im, int x, int y)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idy < y && idx < x) {

		if (idx < x / 2 && idy < y / 2) {
			ore[idy * x + idx] = re[(idy + y / 2) * x + (idx + x / 2)];
			ore[(idy + y / 2) * x + (idx + x / 2)] = re[idy * x + idx];
			oim[idy * x + idx] = im[(idy + y / 2) * x + (idx + x / 2)];
			oim[(idy + y / 2) * x + (idx + x / 2)] = im[idy * x + idx];
		}
		else if (idx >= x / 2 && idy < y / 2) {
			ore[idy * x + idx] = re[(idy + y / 2) * x + (idx - x / 2)];
			ore[(idy + y / 2) * x + (idx - x / 2)] = re[idy * x + idx];
			oim[idy * x + idx] = im[(idy + y / 2) * x + (idx - x / 2)];
			oim[(idy + y / 2) * x + (idx - x / 2)] = im[idy * x + idx];
		}



	}
}

//floatXcufftCom
__global__ void mulcomcufftcom(hipfftComplex* out, float* re, float* im, hipfftComplex* in, int s)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < s) {

		out[idx] = make_hipComplex(re[idx] * hipCrealf(in[idx]) - im[idx] * hipCimagf(in[idx]),
			re[idx] * hipCimagf(in[idx]) + im[idx] * hipCrealf(in[idx]));

	}
}


__global__ void pad_cufftcom2cufftcom(hipfftComplex* out, int lx, int ly, hipfftComplex* in, int sx, int sy)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idx < sx && idy < sy) {
		out[(idy + ly / 4) * lx + (idx + lx / 4)] = in[idy * sx + idx];
	}


}


__global__ void elimpad(hipfftComplex* out, int sx, int sy, hipfftComplex* in, int lx, int ly)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idx < sx && idy < sy) {
		out[idy * sx + idx] = in[(idy + ly / 4) * lx + (idx + lx / 4)];
	}
}





void kaku_cuda(hipfftComplex* devicein, float* ReHs, float* ImHs, int x, int y) {

	hipfftComplex* devpad;
	hipMalloc((void**)&devpad, sizeof(hipfftComplex) * 4 * x * y);
	hipMemset(devpad, 0, sizeof(hipfftComplex) * 4 * x * y);

	dim3 grid(16, 16), block(32, 32);
	pad_cufftcom2cufftcom << <grid, block >> > (devpad, 2 * x, 2 * y, devicein, x, y);


	////�f�o�b�O
	//hipfftComplex* debug;
	//debug = (hipfftComplex*)malloc(sizeof(hipfftComplex) * 4 * x * y);
	//hipMemcpy(debug, devpad, sizeof(hipfftComplex) * 4 * x * y, hipMemcpyDeviceToHost);
	//My_ComArray_2D* test;
	//test = new My_ComArray_2D(4 * x * y, 2 * x, 2 * y);
	//cufftcom_to_mycom(test, debug, 4 * x * y);
	//My_Bmp* timg;
	//timg = new My_Bmp(2 * x, 2 * y);
	//timg->data_to_ucimg(test->Re);
	//string wp = "./pad.bmp";
	//timg->img_write(wp);



	fft_2D_cuda_dev(2 * x, 2 * y, devpad);


	////�f�o�b�O
	//hipMemcpy(debug, devpad, sizeof(hipfftComplex) * 4 * x * y, hipMemcpyDeviceToHost);
	//cufftcom_to_mycom(test, debug, 4 * x * y);
	//My_ComArray_2D* H;
	//H = new My_ComArray_2D(4 * x * y, 2 * x, 2 * y);
	//H->H_kaku(532e-09, 0.1, 1.496e-05);
	//test->mul_complex(H);
	//cout << test->Re[4 * x * y - 500] <<endl<< test->Re[4 * x * y - 100] << endl;



	//�|���Z
	hipfftComplex* rslt;
	hipMalloc((void**)&rslt, sizeof(hipfftComplex) * x * y * 4);
	mulcomcufftcom << <1024, 1024 >> > (rslt, ReHs, ImHs, devpad, 4 * x * y);




 //   //�f�o�b�O
	//hipMemcpy(debug, rslt, sizeof(hipfftComplex) * 4 * x * y, hipMemcpyDeviceToHost);
	//cufftcom_to_mycom(test, debug, 4 * x * y);
	//cout << test->Re[4 * x * y - 500] <<endl<< test->Re[4 * x * y - 100] << endl;


	ifft_2D_cuda_dev(2 * x, 2 * y, rslt);


	////�f�o�b�O
	//hipMemcpy(debug, rslt, sizeof(hipfftComplex) * 4 * x * y, hipMemcpyDeviceToHost);
	//cufftcom_to_mycom(test, debug, 4 * x * y);
	//test->power(test->Re);
	//timg->data_to_ucimg(test->Re);
	//string wp2 = "./beforeelim.bmp";
	//timg->img_write(wp2);



	//devicein��0elim
	dim3 grid2(32, 32), block2(32, 32);
	elimpad<<<grid2, block2>>>(devicein, x, y, rslt, 2 * x, 2 * y);

}


void Hcudaf_shiftf(float* devReH, float* devImH, int x, int y, float d, float z, float lamda) {
	float* ReH, * ImH;
	hipMalloc((void**)&ReH, sizeof(float) * x * y);
	hipMalloc((void**)&ImH, sizeof(float) * x * y);

	float u = 1 / (x * d), v = 1 / (y * d);
	dim3 grid(32, 32), block(32, 32);
	Hcudaf << <grid, block >> > (ReH, ImH, x, y, u, v, z, lamda);
	shiftf << <grid, block >> > (devReH, devImH, ReH, ImH, x, y);

	hipFree(ReH);
	hipFree(ImH);
}


string impath = "./lena512x512.bmp";

float d = 1.496e-5;
float lamda = 532e-09;
float z = 0.1;


int main(void) {
	My_Bmp* img;
	img = new My_Bmp(SX, SY);
	img->img_read(impath);

	My_ComArray_2D* com;
	com = new My_ComArray_2D(SX * SY, SX, SY);
	img->ucimg_to_double(com->Re);

	/*float* debug;
	debug = (float*)malloc(sizeof(float) * SX * SY * 4);*/

	clock_t start = clock();


	//�y�[�W�Œ�ł�OK
	hipfftComplex* host;
	hipHostMalloc((void**)&host, sizeof(hipfftComplex) * SX * SY);
	//host = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SX * SY);
	set_cufftcomplex(host, com->Re, com->Im, SX * SY);

	hipfftComplex* dev;
	hipMalloc((void**)&dev, sizeof(hipfftComplex) * SX * SY);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * SX * SY, hipMemcpyHostToDevice);

	//H���f�o�C�X�Ōv�Z
	/*float* ReH, * ImH;
	hipMalloc((void**)&ReH, sizeof(float) * SX * SY * 4);
	hipMalloc((void**)&ImH, sizeof(float) * SX * SY * 4);*/
	float* ReHs, * ImHs;
	hipMalloc((void**)&ReHs, sizeof(float) * SX * SY * 4);
	hipMalloc((void**)&ImHs, sizeof(float) * SX * SY * 4);
	Hcudaf_shiftf(ReHs, ImHs, 2 * SX, 2 * SY, d, z, lamda);
	/*float u = 1 / (2 * SX * d), v = 1 / (2 * SY * d);
	dim3 grid(32, 32), block(32, 32);
	Hcudaf << <grid, block >> > (ReH, ImH, 2 * SX, 2 * SY, u, v, z, lamda);
	shiftf << <grid, block >> > (ReHs, ImHs, ReH, ImH, 2 * SX, 2 * SY);
	hipFree(ReH);
	hipFree(ImH);*/


	////H�f�o�b�O
	//hipMemcpy(debug, ReHs, sizeof(float) * 4 * SX * SY, hipMemcpyDeviceToHost);
	//cout << debug[2 * SX * SY - 1] <<"\n" << debug[4 * SX * SY - 2] << endl;


	kaku_cuda(dev, ReHs, ImHs, SX, SY);


	hipMemcpy(host, dev, sizeof(hipfftComplex) * SX * SY, hipMemcpyDeviceToHost);
	My_ComArray_2D* out;
	out = new My_ComArray_2D(SX * SY, SX, SY);
	cufftcom_to_mycom(out, host, SX * SY);

	////�f�o�b�O
	//cout <<"real" << out->Re[SX * SY - 1] << "\n" << out->Re[SX * SY - 200] << endl;
	//cout <<"imag" << out->Im[SX * SY - 1] << "\n" << out->Im[SX * SY - 200] << endl;

	out->power(out->Re);
	

	clock_t end = clock();
	cout << "�v�Z����:" << (double)(end - start) << endl;

	////�f�o�b�O
	//cout <<"power" << out->Re[SX * SY - 1] << "\n" << out->Re[SX * SY - 200] << endl;





	/*My_ComArray_2D* test;
	test = new My_ComArray_2D(4 * SX * SY, 2 * SX, 2 * SY);
	test->H_kaku(lamda, z, d);

	cout << test->Re[2 * SX * SY - 1] <<"\n" << test->Re[4 * SX * SY - 2] << endl;*/



	My_Bmp* timg;
	timg = new My_Bmp(SX, SY);
	timg->data_to_ucimg(out->Re);
	string wp = "./cuda.bmp";
	timg->img_write(wp);

	hipFree(dev);
	hipFree(host);
	hipFree(ReHs);
	hipFree(ImHs);
	return 0;
}
