#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

//#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

#ifndef __HIPCC__
#define __HIPCC__
#endif 

#define sqr(x) ((x)*(x))
#define SX 512
#define SY 512


//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)

using namespace std;
using namespace cv;



void set_cufftcomplex(hipfftComplex* cuconp, double* Re, double* Im, int size) {
	for (int i = 0; i < size; i++) {
		cuconp[i] = make_hipComplex((float)Re[i], (float)Im[i]);
	}
}

void set_Re_cufftcomplex(hipfftComplex* cuconp, double* Re, int size) {
	for (int i = 0; i < size; i++) {
		cuconp[i] = make_hipComplex((float)Re[i], 0.0f);
	}
}

__global__ void pad(double* out, int x, int y, double* in)
{
	int X = 2 * x;
	int Y = 2 * y;

	//���͂��ꂽ�摜�f�[�^���O���߂��Ĕ{�̑傫���̉摜�ɂ���
	for (int i = Y / 4; i < y + Y / 4; i++) {
		for (int j = X / 4; j < x + X / 4; j++) {
			out[i * X + j] = in[(i - Y / 4) * x + (j - X / 4)];
		}
	}
}


void fft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
	hipfftHandle plan;
	//double�̂Ƃ���Z2Z?
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
	hipfftDestroy(plan);


}


void ifft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
	hipfftDestroy(plan);
}

void cufftcom_to_mycom(My_ComArray_2D* out, hipfftComplex* in, int s) {
	for (int i = 0; i < s; i++) {
		out->Re[i] = (double)hipCrealf(in[i]);
		out->Im[i] = (double)hipCimagf(in[i]);

	}
}



__global__ void Hcudaf(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
{



	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idy < y && idx < x) {
		Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
		Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
	}


}

__global__ void  shiftf(float* ore, float* oim, float* re, float* im, int x, int y)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idy < y && idx < x) {

		if (idx < x / 2 && idy < y / 2) {
			ore[idy * x + idx] = re[(idy + y / 2) * x + (idx + x / 2)];
			ore[(idy + y / 2) * x + (idx + x / 2)] = re[idy * x + idx];
			oim[idy * x + idx] = im[(idy + y / 2) * x + (idx + x / 2)];
			oim[(idy + y / 2) * x + (idx + x / 2)] = im[idy * x + idx];
		}
		else if (idx >= x / 2 && idy < y / 2) {
			ore[idy * x + idx] = re[(idy + y / 2) * x + (idx - x / 2)];
			ore[(idy + y / 2) * x + (idx - x / 2)] = re[idy * x + idx];
			oim[idy * x + idx] = im[(idy + y / 2) * x + (idx - x / 2)];
			oim[(idy + y / 2) * x + (idx - x / 2)] = im[idy * x + idx];
		}



	}
}

//floatXcufftCom
__global__ void mulcomcufftcom(hipfftComplex* out, float* re, float* im, hipfftComplex* in, int s)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < s) {

		out[idx] = make_hipComplex(re[idx] * hipCrealf(in[idx]) - im[idx] * hipCimagf(in[idx]),
			re[idx] * hipCimagf(in[idx]) + im[idx] * hipCrealf(in[idx]));

	}
}

__global__ void pad_cufftcom(hipfftComplex* out, int lx, int ly, double* Re, double* Im, int sx, int sy)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idx < sx && idy < sy) {
		out[(idy + ly / 4) * lx + (idx + lx / 4)] 
			= make_hipComplex((float)Re[idy * sx + idx], (float)Im[idy * sx + idx]);
	}


}

void kakucuda(My_ComArray_2D* in, int x, int y, float lamda, float d, float z) {

	hipStream_t s1, s2, s3;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);
	hipStreamCreate(&s3);

	double* devRe, * devIm;
	hipMalloc((void**)&devRe, sizeof(double) * x * y);
	hipMalloc((void**)&devIm, sizeof(double) * x * y);





	float* ReH, * ImH;
	hipMalloc((void**)&ReH, sizeof(float) * x * y * 4);
	hipMalloc((void**)&ImH, sizeof(float) * x * y * 4);

	float* ReHs, * ImHs;
	hipMalloc((void**)&ReHs, sizeof(float) * x * y * 4);
	hipMalloc((void**)&ImHs, sizeof(float) * x * y * 4);

	//s1~3�Ń������R�s�[��H�̌v�Z��񓯊����s
	float u = 1 / ((float)2 * x * d), v = 1 / ((float)2 * y * d);
	dim3 grid(32, 32), block(32, 32);
	Hcudaf << <grid, block, 0, s3>> > (ReH, ImH, 2 * x, 2 * y, u, v, z, lamda);
	shiftf << <grid, block, 0, s3>> > (ReHs, ImHs, ReH, ImH, 2 * x, 2 * y);

	hipMemcpyAsync(devRe, in->Re, sizeof(double) * x * y, hipMemcpyHostToDevice, s1);
	hipMemcpyAsync(devIm, in->Im, sizeof(double) * x * y, hipMemcpyHostToDevice, s2);
	
	//�X�g���[���I���
	hipStreamQuery(s1);
	hipStreamQuery(s2);
	hipStreamQuery(s3);


	//pad��cufftcom
	hipfftComplex* devpad;
	hipMalloc((void**)&devpad, sizeof(hipfftComplex) * 4 * x * y);
	hipMemset(devpad, 0, sizeof(hipfftComplex) * 4 * x * y);
	dim3 grid2(16, 16), block2(32, 32);
	pad_cufftcom << <grid2, block2 >> > (devpad, 2 * x, 2 * y, devRe, devIm, x, y);



	fft_2D_cuda_dev(2 * x, 2 * y, devpad);

	


	//�|���Z
	hipfftComplex* rslt;
	hipMalloc((void**)&rslt, sizeof(hipfftComplex) * x * y * 4);
	//dim3 grid3(1024, 1024), block3(1024, 1024);
	mulcomcufftcom<<<1024, 1024>>>(rslt, ReHs, ImHs, devpad, 4 * x * y);


	ifft_2D_cuda_dev(2 * x, 2 * y, rslt);



	hipfftComplex* host;
	host = (hipfftComplex*)malloc(sizeof(hipfftComplex) * x * y * 4);

	/*hipHostMalloc((void**)&host, sizeof())*/
	hipMemcpy(host, rslt, sizeof(hipfftComplex) * x * y * 4, hipMemcpyDeviceToHost);
	My_ComArray_2D* tmp;
	tmp = new My_ComArray_2D(4 * x * y, 2 * x, 2 * y);
	cufftcom_to_mycom(tmp, host, 4 * x * y);

	//in�ɏo��
	tmp->extract_center(in);

	delete tmp;
	free(host);

	hipFree(devpad);
	hipFree(ReH);
	hipFree(ImH);
	hipFree(ReHs);
	hipFree(ImHs);

	hipFree(rslt);
}

__global__ void pad_cufftcom2cufftcom(hipfftComplex* out, int lx, int ly, hipfftComplex* in, int sx, int sy)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idx < sx && idy < sy) {
		out[(idy + ly / 4) * lx + (idx + lx / 4)] = in[idy * sx + idx];
	}


}


__global__ void elimpad(hipfftComplex* out, int sx, int sy, hipfftComplex* in, int lx, int ly)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idx < sx && idy < sy) {
		out[idy * sx + idx] = in[(idy + ly / 4) * lx + (idx + lx / 4)];
	}
}


void kaku_cuda(hipfftComplex* devicein, float* ReHs, float* ImHs, int x, int y) {

	hipfftComplex* devpad;
	hipMalloc((void**)&devpad, sizeof(hipfftComplex) * 4 * x * y);
	hipMemset(devpad, 0, sizeof(hipfftComplex) * 4 * x * y);

	dim3 grid(16, 16), block(32, 32);
	pad_cufftcom2cufftcom<<<grid, block>>>(devpad, 2 * x, 2 * y, devicein, x, y);
	fft_2D_cuda_dev(2 * x, 2 * y, devpad);

	//�|���Z
	hipfftComplex* rslt;
	hipMalloc((void**)&rslt, sizeof(hipfftComplex) * x * y * 4);
	mulcomcufftcom<<<1024, 1024>>>(rslt, ReHs, ImHs, devpad, 4 * x * y);

	ifft_2D_cuda_dev(2 * x, 2 * y, rslt);

	//devicein��0elim
	elimpad<< <grid, block >> > (devicein, x, y, devpad, 2 * x, 2 * y);

}

string impath = "./lena512x512.bmp";
string ompath2 = "./output2.bmp";
string lastpath = "./cudakaku.bmp";
string lastpath2 = "./cudakaku2.bmp";


int main(void) {
	My_Bmp* img;
	img = new My_Bmp(SX, SY);
	img->img_read(impath);





	//GPU,H��kaku�Ōv�Z
	My_ComArray_2D* comcuda;
	comcuda = new My_ComArray_2D(SX * SY, SX, SY);
	img->ucimg_to_double(comcuda->Re);

	clock_t t5 = clock();
	kakucuda(comcuda, SX, SY, (float)532e-09, (float)1.496e-05, (float)0.1);
	clock_t t6 = clock();

	cout << "GPU�v�Z����:" << (double)(t6 - t5) << endl;
	comcuda->power(comcuda->Re);

	My_Bmp* img4;
	img4 = new My_Bmp(SX, SY);
	img4->data_to_ucimg(comcuda->Re);
	img4->img_write(lastpath);







	//GPU,H�͕ʂŌv�Z���ēn��
	My_ComArray_2D* comcuda2;
	comcuda2 = new My_ComArray_2D(SX * SY, SX, SY);
	img->ucimg_to_double(comcuda2->Re);

	clock_t ts = clock();

	hipfftComplex* host;
	//hipHostMalloc((void**)&host, sizeof(hipfftComplex) * SX * SY);
	host = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SX * SY);
	set_cufftcomplex(host, comcuda2->Re, comcuda2->Im, SX * SY);
	hipfftComplex* dev;
	hipMalloc((void**)&dev, sizeof(hipfftComplex) * SX * SY);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * SX * SY, hipMemcpyHostToDevice);

	//H���f�o�C�X�Ōv�Z
	float* ReH, * ImH;
	hipMalloc((void**)&ReH, sizeof(float) * SX * SY * 4);
	hipMalloc((void**)&ImH, sizeof(float) * SX * SY * 4);
	float* ReHs, * ImHs;
	hipMalloc((void**)&ReHs, sizeof(float) * SX * SY * 4);
	hipMalloc((void**)&ImHs, sizeof(float) * SX * SY * 4);
	float u = 1 / ((float)2 * SX * (float)1.496e-05), v = 1 / ((float)2 * SY * (float)1.496e-05);
	dim3 grid(32, 32), block(32, 32);
	Hcudaf << <grid, block>> > (ReH, ImH, 2 * SX, 2 * SY, u, v, (float)0.1, (float)532e-09);
	shiftf << <grid, block>> > (ReHs, ImHs, ReH, ImH, 2 * SX, 2 * SY);
	hipFree(ReH);
	hipFree(ImH);

	kaku_cuda(dev, ReHs, ImHs, SX, SY);
	hipMemcpy(host, dev, sizeof(hipfftComplex) * SX * SY, hipMemcpyDeviceToHost);
	My_ComArray_2D* out;
	out = new My_ComArray_2D(SX * SY, SX, SY);
	cufftcom_to_mycom(out, host, SX * SY);


	clock_t tl = clock();
	cout << "GPU2�v�Z����:" << (double)(tl - ts) << endl;

	out->power(out->Re);
	img4->data_to_ucimg(out->Re);
	img4->img_write(lastpath2);





	//CPU
	My_ComArray_2D* com2;
	com2 = new My_ComArray_2D(SX * SY, SX, SY);
	img->ucimg_to_double(com2->Re);

	clock_t t3 = clock();

	My_ComArray_2D* H2;
	H2 = new My_ComArray_2D(4 * SX * SY, 2 * SX, 2 * SY);
	H2->H_kaku(532e-09, 0.1, 1.496e-05);

	H2->kaku(com2, com2);

	clock_t t4 = clock();
	cout << "CPU�v�Z����:" << (double)(t4 - t3) << endl;


	com2->power(com2->Re);


	My_Bmp* img3;
	img3 = new My_Bmp(SX, SY);
	img3->data_to_ucimg(com2->Re);
	img3->img_write(ompath2);

	delete img;
	delete img4;
	delete comcuda;
	delete com2;
	delete H2;
	delete img3;
	return 0;
}