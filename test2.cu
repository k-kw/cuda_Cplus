#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

//�������C�u�����C���N���[�h
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)

using namespace std;
using namespace cv;

//�p�����[�^
#define BX 28       //bindat��
#define BY 28       //bindat�̏c

//SX,SY�͍��̂Ƃ���2�̊K��̐����`�̂�
#define SX 4096     //SLM�ł̉���f��(4�Ŋ���鐮���Ɍ���)
#define SY 2048   //SLM�ł̏c��f��(4�Ŋ���鐮���Ɍ���)

#define short 1024     //�Z��

#define N 1       //�摜�̖���
#define LENS_SIZE 32 //�g�U�����Y�̃����Y�T�C�Y

#define CHECK_NUM N  //�V�~�����[�V�����摜���`�F�b�N����ԍ�

//#define lam 532e-09  //�g��
//#define d 1.496e-05 //��f�s�b�`
//#define a 0.1 //�`������1
//#define b 0.03 //�`������2
//#define f 0.03 //�œ_����

float lamda = 532e-09;
float d = 3.74e-06;
float a = 0.1;
float b = 0.001;
float f = 0.001;


#define resolution pow(2, 8) //�𑜓x
#define approx false    //�����Y�̎��̋ߎ�


//CUDA
#define sqr(x) ((x)*(x))

#ifndef __HIPCC__
#define __HIPCC__
#endif 

//0���ߌ�摜�T�C�Y
#define SX2 2*SX
#define SY2 2*SY

//1�����̃O���b�h�ƃu���b�N
//���X���b�h��
#define Nthread SX2*SY2
//�u���b�N���̃X���b�h��1=<BS=<1024
#define BS 1024



//�񎟌��̃O���b�h�ƃu���b�N
//�u���b�N������̃X���b�h���͍��v1024�܂łȂ̂ŁAblock(32,32)��葝�₹�Ȃ�
//grid�͏���Ȃ��H
//SX,SY=512,512
//dim3 grid(32, 32), block(32, 32), grid2(16, 16);


//���̂Ƃ���2�̊K��̐����`�ł����ł��Ȃ�,�f�o�b�O�̕K�v����H
//SX,SY=4096,4096
//dim3 grid(256, 256), block(32, 32), grid2(128, 128);

#define blockx 32
#define blocky 32

dim3 grid((SX2 + blockx - 1) / blockx, (SY2 + blocky - 1) / blocky), block(blockx, blocky), grid2((SX + blockx - 1) / blockx, (SY + blocky - 1) / blocky);





//�֐��Q
void set_cufftcomplex(hipfftComplex* cuconp, double* Re, double* Im, int size) {
    for (int i = 0; i < size; i++) {
        cuconp[i] = make_hipComplex((float)Re[i], (float)Im[i]);
    }
}

__global__ void normfft(hipfftComplex* dev, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < x * y) {
        dev[idx] = make_hipComplex(hipCrealf(dev[idx]) / (x * y), hipCimagf(dev[idx]) / (x * y));
    }
}

void fft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
    hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
    hipfftDestroy(plan);


}


void ifft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
    hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
    hipfftDestroy(plan);
}

void cufftcom2mycom(My_ComArray_2D* out, hipfftComplex* in, int s) {
    for (int i = 0; i < s; i++) {
        out->Re[i] = (double)hipCrealf(in[i]);
        out->Im[i] = (double)hipCimagf(in[i]);

    }
}



__global__ void Hcudaf(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    /*if (idy >= y || idx >= x) {
        return;
    }

    Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
    Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));*/
    if (idy < y && idx < x) {
        Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
        Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
    }
}

void Hnotgpu(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
{
    for (int i = 0; i < y; i++) {
        for (int j = 0; j < x; j++) {
            Re[i * x + j] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)j - x / 2)) - sqr(v * ((float)i - y / 2))));
            Im[i * x + j] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)j - x / 2)) - sqr(v * ((float)i - y / 2))));
        }
    }
}


__global__ void  shiftf(float* ore, float* oim, float* re, float* im, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {

        if (idx < x / 2 && idy < y / 2) {
            ore[idy * x + idx] = re[(idy + y / 2) * x + (idx + x / 2)];
            ore[(idy + y / 2) * x + (idx + x / 2)] = re[idy * x + idx];
            oim[idy * x + idx] = im[(idy + y / 2) * x + (idx + x / 2)];
            oim[(idy + y / 2) * x + (idx + x / 2)] = im[idy * x + idx];
        }
        else if (idx >= x / 2 && idy < y / 2) {
            ore[idy * x + idx] = re[(idy + y / 2) * x + (idx - x / 2)];
            ore[(idy + y / 2) * x + (idx - x / 2)] = re[idy * x + idx];
            oim[idy * x + idx] = im[(idy + y / 2) * x + (idx - x / 2)];
            oim[(idy + y / 2) * x + (idx - x / 2)] = im[idy * x + idx];
        }
    }
}

void shiftnotgpu(float* ore, float* oim, float* re, float* im, int x, int y) {

    for (int i = 0; i < y; i++) {
        for (int j = 0; j < x; j++) {
            if (j < x / 2 && i < y / 2) {
                ore[i * x + j] = re[(i + y / 2) * x + (j + x / 2)];
                ore[(i + y / 2) * x + (j + x / 2)] = re[i * x + j];
                oim[i * x + j] = im[(i + y / 2) * x + (j + x / 2)];
                oim[(i + y / 2) * x + (j + x / 2)] = im[i * x + j];
            }
            else if (j >= x / 2 && i < y / 2) {
                ore[i * x + j] = re[(i + y / 2) * x + (j - x / 2)];
                ore[(i + y / 2) * x + (j - x / 2)] = re[i * x + j];
                oim[i * x + j] = im[(i + y / 2) * x + (j - x / 2)];
                oim[(i + y / 2) * x + (j - x / 2)] = im[i * x + j];
            }
        }
    }
}

//floatXcufftCom
__global__ void mulcomcufftcom(hipfftComplex* out, float* re, float* im, hipfftComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex(re[idx] * hipCrealf(in[idx]) - im[idx] * hipCimagf(in[idx]),
            re[idx] * hipCimagf(in[idx]) + im[idx] * hipCrealf(in[idx]));

    }
}


//doubleXcufftCom
__global__ void muldoublecomcufftcom(hipfftComplex* out, double* re, double* im, hipfftComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex((float)re[idx] * hipCrealf(in[idx]) - (float)im[idx] * hipCimagf(in[idx]),
            (float)re[idx] * hipCimagf(in[idx]) + (float)im[idx] * hipCrealf(in[idx]));

    }
}

__global__ void pad_cufftcom2cufftcom(hipfftComplex* out, int lx, int ly, hipfftComplex* in, int sx, int sy)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < sx && idy < sy) {
        out[(idy + ly / 4) * lx + (idx + lx / 4)] = in[idy * sx + idx];
    }

}


__global__ void elimpad(hipfftComplex* out, int sx, int sy, hipfftComplex* in, int lx, int ly)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < sx && idy < sy) {
        out[idy * sx + idx] = in[(idy + ly / 4) * lx + (idx + lx / 4)];
    }
}

//
//
//void kaku_cuda(hipfftComplex* devicein, float* ReHs, float* ImHs, int x, int y, dim3 grid, dim3 block, int mulgrid, int mulblock) {
//
//    hipfftComplex* devpad;
//    hipMalloc((void**)&devpad, sizeof(hipfftComplex) * 4 * x * y);
//    hipMemset(devpad, 0, sizeof(hipfftComplex) * 4 * x * y);
//
//    pad_cufftcom2cufftcom << <grid, block >> > (devpad, 2 * x, 2 * y, devicein, x, y);
//
//
//
//    fft_2D_cuda_dev(2 * x, 2 * y, devpad);
//
//
//    //�|���Z
//    hipfftComplex* rslt;
//    hipMalloc((void**)&rslt, sizeof(hipfftComplex) * x * y * 4);
//    mulcomcufftcom << <mulgrid, mulblock >> > (rslt, ReHs, ImHs, devpad, 4 * x * y);
//
//
//    ifft_2D_cuda_dev(2 * x, 2 * y, rslt);
//
//
//
//    //devicein��0elim
//    elimpad << <grid, block >> > (devicein, x, y, rslt, 2 * x, 2 * y);
//
//
//
//    hipFree(devpad);
//    hipFree(rslt);
//
//}
//
//void kakucuda(hipfftComplex* devin, int inx, int iny, hipfftComplex* devpad, hipfftComplex* mul, float* ReHs, float* ImHs
//    , dim3 gridpadelim, dim3 block, int mulgrid, int mulblock) {
//
//    hipMemset(devpad, 0, sizeof(hipfftComplex) * 4 * inx * iny);
//    pad_cufftcom2cufftcom<<<gridpadelim, block >>>(devpad, 2 * inx, 2 * iny, devin, inx, iny);
//    fft_2D_cuda_dev(2 * inx, 2 * iny, devpad);
//    mulcomcufftcom<<<mulgrid, mulblock >> > (mul, ReHs, ImHs, devpad, 4 * inx * iny);
//    ifft_2D_cuda_dev(2 * inx, 2 * iny, mul);
//    elimpad<<<gridpadelim, block >>>(devin, inx, iny, mul, 2 * inx, 2 * iny);
//}
//

void Hcudaf_shiftf(float* devReH, float* devImH, int x, int y, float d, float z, float lamda, dim3 grid, dim3 block) {
    float* ReH, * ImH;
    hipMalloc((void**)&ReH, sizeof(float) * x * y);
    hipMalloc((void**)&ImH, sizeof(float) * x * y);

    float u = 1 / (x * d), v = 1 / (y * d);

    Hcudaf << <grid, block >> > (ReH, ImH, x, y, u, v, z, lamda);
    shiftf << <grid, block >> > (devReH, devImH, ReH, ImH, x, y);

    hipFree(ReH);
    hipFree(ImH);
}


void Hnotgpushift(float* devReH, float* devImH, int x, int y, float d, float z, float lamda, dim3 grid, dim3 block) {
    /* float* ReH, * ImH;
     hipMalloc((void**)&ReH, sizeof(float) * x * y);
     hipMalloc((void**)&ImH, sizeof(float) * x * y);*/

    float* Re, * Im, * Res, * Ims;
    Re = new float[x * y];
    Im = new float[x * y];
    Res = new float[x * y];
    Ims = new float[x * y];

    float u = 1 / (x * d), v = 1 / (y * d);

    Hnotgpu(Re, Im, x, y, u, v, z, lamda);

    shiftnotgpu(Res, Ims, Re, Im, x, y);

    //hipMemcpy(ReH, Re, sizeof(float) * x * y, hipMemcpyHostToDevice);
    //hipMemcpy(ImH, Im, sizeof(float) * x * y, hipMemcpyHostToDevice);

    //shiftf << <grid, block >> > (devReH, devImH, ReH, ImH, x, y);

    /*hipFree(ReH);
    hipFree(ImH);*/

    hipMemcpy(devReH, Res, sizeof(float) * x * y, hipMemcpyHostToDevice);
    hipMemcpy(devImH, Ims, sizeof(float) * x * y, hipMemcpyHostToDevice);


    delete[]Re;
    delete[]Im;
    delete[]Res;
    delete[]Ims;

}














//�t�@�C���p�X
string binpath = "../../../../dat/bindat/1byte/fm_28_1.dat";
string simpath = "../../../../dat/simdat/SLM_phase/1byte/lsd/test_sim.dat";
string oriimg = "./test.bmp";
string simimg = "./testsim.bmp";
string t = "exp.bmp";

string impath = "./pad.bmp";


#define shx 4096
#define shy 2048

#define size SX*SY
#define pads 4*SX*SY

int main() {
    My_Bmp* img;
    img = new My_Bmp(SX, SY);
    img->img_read(impath);

    


    ////�摜�f�[�^���g�傷��Ƃ�CV_8U�ł��
    ////�摜�f�[�^��cv::Mat�ɃR�s�[
    //Mat bin_mat(SY, SX, CV_8U);
    //memcpy(bin_mat.data, img->img, SX * SY * sizeof(unsigned char));
    //imshow("View", bin_mat);
    //waitKey(0);
   
    ////�g��
    //Mat bin_mat_res(shy, shx, CV_8U);
    //resize(bin_mat, bin_mat_res, Size(shx, shy));
    //bin_mat.release();
    //imshow("View", bin_mat_res);
    //waitKey(0);

    My_ComArray_2D* com;
    com = new My_ComArray_2D(shy * shx, shx, shy);

    /*unsigned char* res;
    res = new unsigned char[shy * shx];

    memcpy(res, bin_mat_res.data, shy * shx * sizeof(unsigned char));*/

    com->data_to_ReIm(img->img);

    My_Bmp* out;
    out = new My_Bmp(shx, shy);
    out->data_to_ucimg(com->Re);
    string r = "./onlyres.bmp";
    out->img_write(r);

    hipfftComplex* host;
    hipHostMalloc((void**)&host, sizeof(hipfftComplex) * shy * shx);
    //host = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SX * SY);
    set_cufftcomplex(host, com->Re, com->Im, shy * shx);


    hipfftComplex* dev;
    hipMalloc((void**)&dev, sizeof(hipfftComplex) * shx * shy);
    hipMemcpy(dev, host, sizeof(hipfftComplex) * shx * shy, hipMemcpyHostToDevice);

    fft_2D_cuda_dev(shx, shy, dev);

    hipMemcpy(host, dev, sizeof(hipfftComplex) * shx * shy, hipMemcpyDeviceToHost);
    cufftcom2mycom(com, host, shx * shy);

    My_ComArray_2D* H;
    H = new My_ComArray_2D(shx * shy, shx, shy);
    H->H_kaku((double)lamda, (double)a, (double)d);

    H->mul_complex(com);
    set_cufftcomplex(host, H->Re, H->Im, shx * shy);
    hipMemcpy(dev, host, sizeof(hipfftComplex) * shx * shy, hipMemcpyHostToDevice);

    /*com->power(com->Re);



    out->data_to_ucimg(com->Re);
    string fft = "./fftlena.bmp";
    out->img_write(fft);*/


    ifft_2D_cuda_dev(shx, shy, dev);

    hipMemcpy(host, dev, sizeof(hipfftComplex) * shx * shy, hipMemcpyDeviceToHost);

    cufftcom2mycom(com, host, shx * shy);

    com->power(com->Re);


   
    out->data_to_ucimg(com->Re);
    string test = "./restore_rec.bmp";
    out->img_write(test);




    //float* Re, * Im;
    //Re = new float[pads];
    //Im = new float[pads];


    //float* Re2, * Im2;
    //Re2 = new float[pads];
    //Im2 = new float[pads];



    ////H�z����f�o�C�X���ō쐬
    //float* ReHa, * ImHa;
    //hipMalloc((void**)&ReHa, sizeof(float) * pads);
    //hipMalloc((void**)&ImHa, sizeof(float) * pads);





    ////�u���b�N������̃X���b�h���͍��v1024�܂łȂ̂ŁAblock(32,32)��葝�₹�Ȃ�
    ////grid�͏���Ȃ��H
    //Hcudaf_shiftf(ReHa, ImHa, SX2, SY2, d, a, lamda, grid, block);
    //
    //hipMemcpy(Re, ReHa, sizeof(float) * pads, hipMemcpyDeviceToHost);
    //hipMemcpy(Im, ImHa, sizeof(float) * pads, hipMemcpyDeviceToHost);

    ////cout << Re[SX] << "\t" << Im[SX] << "\n";

    //

    //Hnotgpushift(ReHa, ImHa, SX2, SY2, d, a, lamda, grid, block);


    //hipMemcpy(Re2, ReHa, sizeof(float) * pads, hipMemcpyDeviceToHost);
    //hipMemcpy(Im2, ImHa, sizeof(float) * pads, hipMemcpyDeviceToHost);

    ////cout << Re[SX] << "\t" << Im[SX] << "\n";


    //My_ComArray_2D H(SX2* SY2, SX2, SY2);
    //H.H_kaku((double)lamda, (double)a, (double)d);


    //for (int i = 0; i < SY2; i++) {
    //    for (int j = 0; j < SX2; j++) {
    //        cout << Re[i * SX2 + j] << "\t" << Re2[i * SX2 + j] <<"\t" << H.Re[i * SX2 + j] << "\n";
    //    }
    //}

    ////float* ReHb, * ImHb;
    ////hipMalloc((void**)&ReHb, sizeof(float) * pads);
    ////hipMalloc((void**)&ImHb, sizeof(float) * pads);
    //////�u���b�N������̃X���b�h���͍��v1024�܂łȂ̂ŁAblock(32,32)��葝�₹�Ȃ�
    //////grid�͏���Ȃ��H
    ////Hcudaf_shiftf(ReHb, ImHb, 2 * shx, 2 * shy, d, b, lamda, grid, block);

    ////Hnotgpushift(ReHb, ImHb, 2 * shx, 2 * shy, d, b, lamda, grid, block);





















    return 0;
}