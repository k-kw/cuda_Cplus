#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

//#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

#ifndef __HIPCC__
#define __HIPCC__
#endif 

#define sqr(x) ((x)*(x))
#define SX 512
#define SY 512


//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)

using namespace std;
using namespace cv;

__global__ void assign2D(int* d_a, int w, int h, int value)
{
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	d_a[idy * w + idx] = value;
}

int main() {
	int w = 1024, h = 1024;
	int* h_a;
	h_a = new int[w * h];

	int* d_a;

	hipMalloc((void**)&d_a, sizeof(int) * w * h);
	hipMemcpy(d_a, h_a, sizeof(int) * w * h, hipMemcpyHostToDevice);

	assign2D << <dim3(64, 64), dim3(16, 16) >> > (d_a, w, h, 5);

	hipMemcpy(h_a, d_a, sizeof(int) * w * h, hipMemcpyDeviceToHost);

	cout << h_a[0] << "\n" << h_a[1];


	delete[]h_a;
	hipFree(d_a);

	return 0;


}