#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

//#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

#ifndef __HIPCC__
#define __HIPCC__
#endif 

#define sqr(x) ((x)*(x))
#define SX 512
#define SY 512


//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)

using namespace std;
using namespace cv;



void set_cufftcomplex(hipfftComplex* cuconp, double* Re, double* Im, int size) {
	for (int i = 0; i < size; i++) {
		cuconp[i] = make_hipComplex((float)Re[i], (float)Im[i]);
	}
}

void set_Re_cufftcomplex(hipfftComplex* cuconp, double* Re, int size) {
	for (int i = 0; i < size; i++) {
		cuconp[i] = make_hipComplex((float)Re[i], 0.0f);
	}
}

//CUDA_FFT2D
void fft_2D_cuda(int x, int y, hipfftComplex* host) {
	hipfftHandle plan;

	hipfftComplex* dev;

	hipMalloc((void**)&dev, sizeof(hipfftComplex) * x * y);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * x * y, hipMemcpyHostToDevice);

	//�t�[���G�ϊ�
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);

	hipMemcpy(host, dev, sizeof(hipfftComplex) * x * y, hipMemcpyDeviceToHost);


	hipFree(dev);
	hipfftDestroy(plan);
}

//CUDA_IFFT2D
void ifft_2D_cuda(int x, int y, hipfftComplex* host) {
	hipfftHandle plan;

	hipfftComplex* dev;

	hipMalloc((void**)&dev, sizeof(hipfftComplex) * x * y);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * x * y, hipMemcpyHostToDevice);

	//�t�[���G�ϊ�
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);

	hipMemcpy(host, dev, sizeof(hipfftComplex) * x * y, hipMemcpyDeviceToHost);

	hipFree(dev);
	hipfftDestroy(plan);
}

__global__ void pad(double* out, int x, int y, double* in)
{
	int X = 2 * x;
	int Y = 2 * y;

	//���͂��ꂽ�摜�f�[�^���O���߂��Ĕ{�̑傫���̉摜�ɂ���
	for (int i = Y / 4; i < y + Y / 4; i++) {
		for (int j = X / 4; j < x + X / 4; j++) {
			out[i * X + j] = in[(i - Y / 4) * x + (j - X / 4)];
		}
	}
}



//__global__ void H(hipfftComplex* H, double lam, double z, double d, int x, int y) {
//	double u = 1 / ((double)x * d), v = 1 / ((double)y * d);
//
//	int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	int idy = blockDim.y * blockIdx.y + threadIdx.y;
//	
//	H[idx] = make_hipComplex((float)Re[i], (float)Im[i]);
//}



//�p�X�y�N�g���@��H�𒼐ڌv�Z����֐�
void H_kaku(My_ComArray_2D* H, double lam, double z, double d, int x, int y) {

	My_ComArray_2D* tmp;
	tmp = new My_ComArray_2D(x * y, x, y);

	double u = 1 / ((double)x * d), v = 1 / ((double)y * d);
	//H�v�Z
	for (int i = 0; i < y; i++) {
		for (int j = 0; j < x; j++) {
			tmp->Re[i * x + j] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((double)j - x / 2)) - sqr(v * ((double)i - y / 2))));
			tmp->Im[i * x + j] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((double)j - x / 2)) - sqr(v * ((double)i - y / 2))));
		}
	}
	//H�V�t�g
	for (int i = 0; i < y; i++) {
		for (int j = 0; j < x; j++) {
			if (j < x / 2 && i < y / 2) {
				H->Re[i * x + j] = tmp->Re[(i + y / 2) * x + (j + x / 2)];
				H->Re[(i + y / 2) * x + (j + x / 2)] = tmp->Re[i * x + j];
				H->Im[i * x + j] = tmp->Im[(i + y / 2) * x + (j + x / 2)];
				H->Im[(i + y / 2) * x + (j + x / 2)] = tmp->Im[i * x + j];
			}
			else if (j >= x / 2 && i < y / 2) {
				H->Re[i * x + j] = tmp->Re[(i + y / 2) * x + (j - x / 2)];
				H->Re[(i + y / 2) * x + (j - x / 2)] = tmp->Re[i * x + j];
				H->Im[i * x + j] = tmp->Im[(i + y / 2) * x + (j - x / 2)];
				H->Im[(i + y / 2) * x + (j - x / 2)] = tmp->Im[i * x + j];
			}
		}
	}

	delete tmp;
};


//2D�摜��0pad�֐�(�c�����ꂼ��Q�{�ɂ���0���߁Ain��out�̓T�C�Y�Ⴄ)
void Opad(double* img_out, int x, int y, double* img_in) {
	int X, Y;
	X = 2 * x;
	Y = 2 * y;

	double* img_tmp;
	img_tmp = new double[X * Y];

	for (int i = 0; i < X * Y; i++) {
		img_tmp[i] = 0;
	}

	//���͂��ꂽ�摜�f�[�^���O���߂��Ĕ{�̑傫���̉摜�ɂ���
	for (int i = Y / 4; i < y + Y / 4; i++) {
		for (int j = X / 4; j < x + X / 4; j++) {
			img_tmp[i * X + j] = img_in[(i - Y / 4) * x + (j - X / 4)];
		}
	}

	for (int i = 0; i < X * Y; i++) {
		img_out[i] = img_tmp[i];
	}

	delete[]img_tmp;

}

void fft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
	hipfftDestroy(plan);

	/*for (int i = 0; i < x * y; i++) {
		dev[i] = make_hipComplex((float)(hipCrealf(dev[i]) / (x * y)), (float)(hipCimagf(dev[i]) / (x * y)));
	}*/
}


void ifft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
	hipfftDestroy(plan);
}

void cufftcom_to_mycom(My_ComArray_2D* out, hipfftComplex* in, int s) {
	for (int i = 0; i < s; i++) {
		out->Re[i] = (double)hipCrealf(in[i]);
		out->Im[i] = (double)hipCimagf(in[i]);

	}
}

//���f���z���Z�֐�
void mul_com(int size, My_ComArray_2D* in1, My_ComArray_2D* in2, My_ComArray_2D* out) {
	double* Retmp, * Imtmp;
	Retmp = new double[size];
	Imtmp = new double[size];

	for (int i = 0; i < size; i++) {
		Retmp[i] = in1->Re[i] * in2->Re[i] - in1->Im[i] * in2->Im[i];
		Imtmp[i] = in1->Re[i] * in2->Im[i] + in1->Im[i] * in2->Re[i];
	}

	for (int i = 0; i < size; i++) {
		out->Re[i] = Retmp[i];
		out->Im[i] = Imtmp[i];
	}
	delete[]Retmp;
	delete[]Imtmp;
};

void elim(My_ComArray_2D* in, int ix, int iy, My_ComArray_2D* out) {
	int x, y;
	x = ix / 2;
	y = iy / 2;

	for (int i = iy / 4; i < y + iy / 4; i++) {
		for (int j = ix / 4; j < x + ix / 4; j++) {
			out->Re[(i - iy / 4) * x + (j - ix / 4)] = in->Re[i * ix + j];
			out->Im[(i - iy / 4) * x + (j - ix / 4)] = in->Im[i * ix + j];

		}
	}

}


__global__ void Hcuda(double* Re, double* Im, int x, int y, double u, double v, double z, double lam)
{
	

	////H�v�Z
	//for (int i = 0; i < y; i++) {
	//	for (int j = 0; j < x; j++) {
	//		H->Re[i * x + j] = cos(2 * M_PI * z * sqrt((1 / lam) * (1 / lam) - (u * ((double)j - x / 2)) * (u * ((double)j - x / 2)) - (v * ((double)i - y / 2)) * (v * ((double)i - y / 2))));
	//		H->Im[i * x + j] = sin(2 * M_PI * z * sqrt((1 / lam) * (1 / lam) - (u * ((double)j - x / 2)) * (u * ((double)j - x / 2)) - (v * ((double)i - y / 2)) * (v * ((double)i - y / 2))));
	//	}
	//}

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idy < y && idx < x) {
		Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((double)idx - x / 2)) - sqr(v * ((double)idy - y / 2))));
		Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((double)idx - x / 2)) - sqr(v * ((double)idy - y / 2))));
	}


}

//
//__global__ void Hcuda(double* Re, double* Im, double d, double lam, double z, int x, int y)
//{
//	double u = 1 / ((double)x * d), v = 1 / ((double)y * d);
//
//
//	int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	int idy = blockDim.y * blockIdx.y + threadIdx.y;
//
//	if (idy < y && idx < x) {
//		Re[idy * x + idx] = (double)3.14;
//		Im[idy * x + idx] = (double)3.14;
//
//	}
//}
//
//__global__ void Hcuda2(double* Re, double* Im, double d, double lam, double z, int x, int y, double value)
//{
//	//double u = 1 / ((double)x * d), v = 1 / ((double)y * d);
//
//
//	int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	int idy = blockDim.y * blockIdx.y + threadIdx.y;
//	if (idy < y && idx < x) {
//		Re[idy * x + idx] = value;
//		Im[idy * x + idx] = value;
//	}
//}
//

__global__ void  shift(double* ore, double* oim, double* re, double* im, int x, int y)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idy < y && idx < x) {

		if (idx < x / 2 && idy < y / 2) {
			ore[idy * x + idx] = re[(idy + y / 2) * x + (idx + x / 2)];
			ore[(idy + y / 2) * x + (idx + x / 2)] = re[idy * x + idx];
			oim[idy * x + idx] = im[(idy + y / 2) * x + (idx + x / 2)];
			oim[(idy + y / 2) * x + (idx + x / 2)] = im[idy * x + idx];
		}
		else if (idx >= x / 2 && idy < y / 2) {
			ore[idy * x + idx] = re[(idy + y / 2) * x + (idx - x / 2)];
			ore[(idy + y / 2) * x + (idx - x / 2)] = re[idy * x + idx];
			oim[idy * x + idx] = im[(idy + y / 2) * x + (idx - x / 2)];
			oim[(idy + y / 2) * x + (idx - x / 2)] = im[idy * x + idx];
		}



	}
}

//void shift(hipfftComplex* out, My_ComArray_2D* in, int x, int y)
//{
//
//	float tmpRe, tmpIm;
//
//	for (int i = 0; i < y; i++) {
//		for (int j = 0; j < x; j++) {
//
//			if (j < x / 2 && i < y / 2) {
//
//				tmpRe = (float)in->Re[(i + y / 2) * x + (j + x / 2)];
//				tmpIm = (float)in->Im[(i + y / 2) * x + (j + x / 2)];
//				out[i * x + j] = make_hipComplex(tmpRe, tmpIm);
//
//				tmpRe = (float)in->Re[i * x + j];
//				tmpIm = (float)in->Im[i * x + j];
//				out[(i + y / 2) * x + (j + x / 2)] = make_hipComplex(tmpRe, tmpIm);
//			}
//
//			else if (j >= x / 2 && i < y / 2) {
//
//				tmpRe = (float)in->Re[(i + y / 2) * x + (j - x / 2)];
//				tmpIm = (float)in->Im[(i + y / 2) * x + (j - x / 2)];
//				out[i * x + j] = make_hipComplex(tmpRe, tmpIm);
//
//				tmpRe = (float)in->Re[i * x + j];
//				tmpIm = (float)in->Im[i * x + j];
//				out[(i + y / 2) * x + (j + x / 2)] = make_hipComplex(tmpRe, tmpIm);
//			}
//		}
//	}
//}



//__global__ void mul_cucom(hipfftComplex* out, hipfftComplex* in1, hipfftComplex* in2, int s)
//{
//	//int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	float tmpRe, tmpIm;
//	/*if (idx < s) {
//		tmpRe = hipCrealf(in1[idx]) * hipCrealf(in2[idx]) - hipCimagf(in1[idx]) * hipCimagf(in2[idx]);
//		tmpIm = hipCrealf(in1[idx]) * hipCimagf(in2[idx]) + hipCimagf(in1[idx]) * hipCrealf(in2[idx]);
//
//		out[idx] = make_hipComplex(tmpRe, tmpIm);
//
//	}*/
//
//	for (int i = 0; i < s; i++) {
//		tmpRe = (hipCrealf(in1[i]) * hipCrealf(in2[i])) - (hipCimagf(in1[i]) * hipCimagf(in2[i]));
//		tmpIm = (hipCrealf(in1[i]) * hipCimagf(in2[i])) + (hipCimagf(in1[i]) * hipCrealf(in2[i]));
//
//		out[i] = make_hipComplex(tmpRe, tmpIm);
//	}
//}

__global__ void mulcom(double* ore, double* oim, double* re, double* im, double* re2, double* im2, int s)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < s) {
		ore[idx] = re[idx] * re2[idx] - im[idx] * im2[idx];
		oim[idx] = re[idx] * im2[idx] + im[idx] * re2[idx];

	}
}

void kaku(My_ComArray_2D* in, int x, int y, double lamda, double d, double z)
{

	My_ComArray_2D* tmp;
	tmp = new My_ComArray_2D(4 * x * y, 2 * x, 2 * y);
	in->zeropad(tmp);

	hipfftComplex* host;
	host = (hipfftComplex*)malloc(sizeof(hipfftComplex) * x * y * 4);
	set_cufftcomplex(host, tmp->Re, tmp->Im, x * y * 4);


	hipfftComplex* dev;
	hipMalloc((void**)&dev, sizeof(hipfftComplex) * x * y * 4);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * x * y * 4, hipMemcpyHostToDevice);


	fft_2D_cuda_dev(2 * x, 2 * y, dev);
	hipMemcpy(host, dev, sizeof(hipfftComplex) * x * y * 4, hipMemcpyDeviceToHost);
	cufftcom_to_mycom(tmp, host, 4 * x * y);
	
	double* re;
	hipMalloc((void**)&re, sizeof(double) * x * y * 4);
	hipMemcpy(re, tmp->Re, sizeof(double) * x * y * 4, hipMemcpyHostToDevice);
	double* im;
	hipMalloc((void**)&im, sizeof(double) * x * y * 4);
	hipMemcpy(im, tmp->Im, sizeof(double) * x * y * 4, hipMemcpyHostToDevice);





	double* ReH, * ImH;
	hipMalloc((void**)&ReH, sizeof(double) * x * y * 4);
	hipMalloc((void**)&ImH, sizeof(double) * x * y * 4);

	double* ReHs, * ImHs;
	hipMalloc((void**)&ReHs, sizeof(double) * x * y * 4);
	hipMalloc((void**)&ImHs, sizeof(double) * x * y * 4);

	double u = 1 / ((double)2 * SX * d), v = 1 / ((double)2 * SY * d);
	dim3 grid(32, 32), block(32, 32);
	Hcuda<<<grid, block>>>(ReH, ImH, 2 * SX, 2 * SY, u, v, z, lamda);
	shift<<<grid, block>>>(ReHs, ImHs, ReH, ImH, 2 * x, 2 * y);



	////�f�o�b�O
	//hipMemcpy(tmp->Re, ReHs, sizeof(double) * x * y * 4, hipMemcpyDeviceToHost);
	//cout << "CUDA\t" << tmp->Re[0] << "\t" << tmp->Re[4 * SX * SY - 1] << endl;
	//My_ComArray_2D* th;
	//th = new My_ComArray_2D(4 * x * y, 2 * x, 2 * y);
	////H_kaku(th, lamda, z, d, 2 * x, 2 * y);
	//th->H_kaku(lamda, z, d);
	//cout << "contemporary\t" << th->Re[0] << "\t" << th->Re[4 * SX * SY - 1] << endl;


	//H_kaku(H, lamda, z, d, 2 * x, 2 * y);
	////H->H_kaku(lamda, z, d);

	/*double* re2;
	hipMalloc((void**)&re2, sizeof(double) * x * y * 4);
	hipMemcpy(re2, H->Re, sizeof(double) * x * y * 4, hipMemcpyHostToDevice);
	double* im2;
	hipMalloc((void**)&im2, sizeof(double) * x * y * 4);
	hipMemcpy(im2, H->Im, sizeof(double) * x * y * 4, hipMemcpyHostToDevice);*/


	//�|���Z
	double* re3;
	hipMalloc((void**)&re3, sizeof(double) * x * y * 4);
	double* im3;
	hipMalloc((void**)&im3, sizeof(double) * x * y * 4);
	
	//dim3 grid(128, 128), block(128, 128);
	mulcom<<<1024,1024>>>(re3, im3, re, im, ReHs, ImHs, x * y * 4);

	hipMemcpy(tmp->Re, re3, sizeof(double) * x * y * 4, hipMemcpyDeviceToHost);
	hipMemcpy(tmp->Im, im3, sizeof(double) * x * y * 4, hipMemcpyDeviceToHost);

	set_cufftcomplex(host, tmp->Re, tmp->Im, x * y * 4);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * x * y * 4, hipMemcpyHostToDevice);



	ifft_2D_cuda_dev(2 * x, 2 * y, dev);


	hipMemcpy(host, dev, sizeof(hipfftComplex) * x * y * 4, hipMemcpyDeviceToHost);
	cufftcom_to_mycom(tmp, host, 4 * x * y);

	//in�ɏo��
	elim(tmp, 2 * x, 2 * y, in);


	free(host);
	hipFree(dev);
	hipFree(re);
	hipFree(im);
	hipFree(ReH);
	hipFree(ImH);
	hipFree(ReHs);
	hipFree(ImHs);
	hipFree(re3);
	hipFree(im3);

	
	delete tmp;

}


void kakucuda(My_ComArray_2D* in, int x, int y, double lamda, double d, double z) {

	My_ComArray_2D* tmp;
	tmp = new My_ComArray_2D(4 * x * y, 2 * x, 2 * y);
	in->zeropad(tmp);

	hipfftDoubleComplex* host;
	host = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex) * x * y * 4);
	set_cufftdoublecomplex(host, tmp->Re, tmp->Im, x * y * 4);


	hipfftComplex* dev;
	hipMalloc((void**)&dev, sizeof(hipfftComplex) * x * y * 4);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * x * y * 4, hipMemcpyHostToDevice);

}

string impath = "./lena512x512.bmp";
string ompath = "./output.bmp";
string ompath2 = "./output2.bmp";

int main(void) {
	My_Bmp* img;
	img = new My_Bmp(SX, SY);
	img->img_read(impath);

	My_ComArray_2D* com;
	com = new My_ComArray_2D(SX * SY, SX, SY);

	img->ucimg_to_double(com->Re);

	clock_t t1 = clock();

	kaku(com, SX, SY, 532e-09, 1.496e-05, 0.1);

	clock_t t2 = clock();
	cout << "�v�Z����:" << (double)(t2 - t1) << endl;
	com->power(com->Re);

	My_Bmp* img2;
	img2 = new My_Bmp(SX, SY);
	img2->data_to_ucimg(com->Re);
	img2->img_write(ompath);





	//My_ComArray_2D* tmp;
	//tmp = new My_ComArray_2D(4 * SX * SY, 2 * SX, 2 * SY);
	////�f�o�b�O
	//double* ReH, * ImH;
	//hipMalloc((void**)&ReH, sizeof(double) * SX * SY * 4);
	//hipMalloc((void**)&ImH, sizeof(double) * SX * SY * 4);


	//dim3 grid(32, 32), block(32, 32);
	//Hcuda2<<<grid, block>>>(ReH, ImH, 1.496e-05, 532e-09, 0.1, 2 * SX, 2 * SY, 3.14);


	//hipMemcpy(tmp->Re, ReH, sizeof(double) * SX * SY * 4, hipMemcpyDeviceToHost);
	//cout << tmp->Re[SX * SY * 4 - 1] << "\t" << tmp->Re[1] << endl;






	My_ComArray_2D* com2;
	com2 = new My_ComArray_2D(SX * SY, SX, SY);
	img->ucimg_to_double(com2->Re);

	clock_t t3 = clock();

	My_ComArray_2D* H2;
	H2 = new My_ComArray_2D(4 * SX * SY, 2 * SX, 2 * SY);
	H2->H_kaku(532e-09, 0.1, 1.496e-05);

	H2->kaku(com2, com2);

	clock_t t4 = clock();
	cout << "�v�Z����:" << (double)(t4 - t3) << endl;


	com2->power(com2->Re);


	My_Bmp* img3;
	img3 = new My_Bmp(SX, SY);
	img3->data_to_ucimg(com2->Re);
	img3->img_write(ompath2);

	delete com;
	delete img;
	delete img2;
	delete com2;
	delete img3;
	return 0;
}