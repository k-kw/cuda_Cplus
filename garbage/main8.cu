#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

//�������C�u�����C���N���[�h
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


//#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

#ifndef __HIPCC__
#define __HIPCC__
#endif 



//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)


#define sqr(x) ((x)*(x))
#define SX 512
#define SY 512

#define SX2 2*SX
#define SY2 2*SY


//���X���b�h��
#define Nthread SX2*SY2

//�u���b�N���̃X���b�h��1=<BS=<1024
#define BS 1024



using namespace std;
using namespace cv;

void set_cufftcomplex(hipfftComplex* cuconp, double* Re, double* Im, int size) {
	for (int i = 0; i < size; i++) {
		cuconp[i] = make_hipComplex((float)Re[i], (float)Im[i]);
	}
}


void fft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
	hipfftDestroy(plan);


}


void ifft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
	hipfftHandle plan;
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
	hipfftDestroy(plan);
}

void cufftcom_to_mycom(My_ComArray_2D* out, hipfftComplex* in, int s) {
	for (int i = 0; i < s; i++) {
		out->Re[i] = (double)hipCrealf(in[i]);
		out->Im[i] = (double)hipCimagf(in[i]);

	}
}



__global__ void Hcudaf(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
{



	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idy < y && idx < x) {
		Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
		Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
	}


}

__global__ void  shiftf(float* ore, float* oim, float* re, float* im, int x, int y)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idy < y && idx < x) {

		if (idx < x / 2 && idy < y / 2) {
			ore[idy * x + idx] = re[(idy + y / 2) * x + (idx + x / 2)];
			ore[(idy + y / 2) * x + (idx + x / 2)] = re[idy * x + idx];
			oim[idy * x + idx] = im[(idy + y / 2) * x + (idx + x / 2)];
			oim[(idy + y / 2) * x + (idx + x / 2)] = im[idy * x + idx];
		}
		else if (idx >= x / 2 && idy < y / 2) {
			ore[idy * x + idx] = re[(idy + y / 2) * x + (idx - x / 2)];
			ore[(idy + y / 2) * x + (idx - x / 2)] = re[idy * x + idx];
			oim[idy * x + idx] = im[(idy + y / 2) * x + (idx - x / 2)];
			oim[(idy + y / 2) * x + (idx - x / 2)] = im[idy * x + idx];
		}



	}
}

//floatXcufftCom
__global__ void mulcomcufftcom(hipfftComplex* out, float* re, float* im, hipfftComplex* in, int s)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < s) {

		out[idx] = make_hipComplex(re[idx] * hipCrealf(in[idx]) - im[idx] * hipCimagf(in[idx]),
			re[idx] * hipCimagf(in[idx]) + im[idx] * hipCrealf(in[idx]));

	}
}


__global__ void pad_cufftcom2cufftcom(hipfftComplex* out, int lx, int ly, hipfftComplex* in, int sx, int sy)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idx < sx && idy < sy) {
		out[(idy + ly / 4) * lx + (idx + lx / 4)] = in[idy * sx + idx];
	}


}


__global__ void elimpad(hipfftComplex* out, int sx, int sy, hipfftComplex* in, int lx, int ly)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;

	if (idx < sx && idy < sy) {
		out[idy * sx + idx] = in[(idy + ly / 4) * lx + (idx + lx / 4)];
	}
}


//
//__global__ void randomdeffuser(unsigned long long seed, float* device, int size)
//{
//	int idx = blockDim.x * blockIdx.x + threadIdx.x;
//	hiprandGenerator_t gen;
//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
//
//	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
//
//	hiprandGenerateUniform(gen, device, size);
//
//	if (idx < size) {
//		device[idx] *= 2 * M_PI;
//	}
//
//	hiprandDestroyGenerator(gen);
//
//}




void kaku_cuda(hipfftComplex* devicein, float* ReHs, float* ImHs, int x, int y,
	dim3 grid, dim3 block, int mulgrid, int mulblock) {

	hipfftComplex* devpad;
	hipMalloc((void**)&devpad, sizeof(hipfftComplex) * 4 * x * y);
	hipMemset(devpad, 0, sizeof(hipfftComplex) * 4 * x * y);

	pad_cufftcom2cufftcom <<<grid, block>>>(devpad, 2 * x, 2 * y, devicein, x, y);



	fft_2D_cuda_dev(2 * x, 2 * y, devpad);


	//�|���Z
	hipfftComplex* rslt;
	hipMalloc((void**)&rslt, sizeof(hipfftComplex) * x * y * 4);
	mulcomcufftcom<<<mulgrid, mulblock>>>(rslt, ReHs, ImHs, devpad, 4 * x * y);


	ifft_2D_cuda_dev(2 * x, 2 * y, rslt);



	//devicein��0elim
	elimpad<<<grid, block>>>(devicein, x, y, rslt, 2 * x, 2 * y);



	hipFree(devpad);
	hipFree(rslt);

}


void Hcudaf_shiftf(float* devReH, float* devImH, int x, int y, float d, float z, float lamda, dim3 grid, dim3 block) {
	float* ReH, * ImH;
	hipMalloc((void**)&ReH, sizeof(float) * x * y);
	hipMalloc((void**)&ImH, sizeof(float) * x * y);

	float u = 1 / (x * d), v = 1 / (y * d);
	
	Hcudaf << <grid, block >> > (ReH, ImH, x, y, u, v, z, lamda);
	shiftf << <grid, block >> > (devReH, devImH, ReH, ImH, x, y);

	hipFree(ReH);
	hipFree(ImH);
}


string impath = "./lena512x512.bmp";

float d = 1.496e-5;
float lamda = 532e-09;
float z = 0.1;
float f = 0.05;
float a = 0.1;



int main(void) {
	My_Bmp* img;
	img = new My_Bmp(SX, SY);
	img->img_read(impath);

	My_ComArray_2D* com;
	com = new My_ComArray_2D(SX * SY, SX, SY);
	img->ucimg_to_double(com->Re);


	clock_t start = clock();


	//�y�[�W�Œ�ł�OK
	hipfftComplex* host;
	hipHostMalloc((void**)&host, sizeof(hipfftComplex) * SX * SY);
	//host = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SX * SY);
	set_cufftcomplex(host, com->Re, com->Im, SX * SY);

	hipfftComplex* dev;
	hipMalloc((void**)&dev, sizeof(hipfftComplex) * SX * SY);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * SX * SY, hipMemcpyHostToDevice);



	//H���f�o�C�X�Ōv�Z
	float* ReHs, * ImHs;
	hipMalloc((void**)&ReHs, sizeof(float) * SX * SY * 4);
	hipMalloc((void**)&ImHs, sizeof(float) * SX * SY * 4);

	//�u���b�N������̃X���b�h���͍��v1024�܂łȂ̂ŁAblock(32,32)��葝�₹�Ȃ�
	//grid�͏���Ȃ��H
	dim3 grid(32, 32), block(32, 32), grid2(16, 16);
	Hcudaf_shiftf(ReHs, ImHs, 2 * SX, 2 * SY, d, z, lamda, grid, block);
	
	//�p�X�y�N�g��
	kaku_cuda(dev, ReHs, ImHs, SX, SY, grid2, block, (Nthread + BS - 1) / BS, BS);


	hipMemcpy(host, dev, sizeof(hipfftComplex) * SX * SY, hipMemcpyDeviceToHost);
	My_ComArray_2D* out;
	out = new My_ComArray_2D(SX * SY, SX, SY);
	cufftcom_to_mycom(out, host, SX * SY);

	
	//�U��
	out->power(out->Re);


	clock_t end = clock();
	cout << "�v�Z����:" << (double)(end - start) << endl;



	////�f�o�b�O
	//float* random;
	//hipMalloc((void**)&random, sizeof(float) * SX * SY * 4);
	//randomdeffuser<<<(Nthread + BS - 1) / BS, BS >>>(5555ULL, random, SX * SY * 4);
	//float* randhost;
	//randhost = (float*)malloc(sizeof(float) * SX * SY * 4);
	//hipMemcpy(randhost, random, sizeof(float) * SX * SY * 4, hipMemcpyDeviceToHost);
	//My_Bmp* randimg;
	//randimg = new My_Bmp(2 * SX, 2 * SY);
	//randimg->data_to_ucimg(randhost);
	//string rp = "./rand.bmp";
	//randimg->img_write(rp);



	My_Bmp* timg;
	timg = new My_Bmp(SX, SY);
	timg->data_to_ucimg(out->Re);
	string wp = "./cuda.bmp";
	timg->img_write(wp);

	hipFree(dev);
	hipFree(host);
	hipFree(ReHs);
	hipFree(ImHs);
	return 0;
}
