#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)

using namespace std;
using namespace cv;

//�p�����[�^1
#define BX 28       //bindat��
#define BY 28       //bindat�̏c

#define SX 540     //SLM�ł̉���f��(4�Ŋ���鐮���Ɍ���)
#define SY 540     //SLM�ł̏c��f��(4�Ŋ���鐮���Ɍ���)
//#define PJRSX 500     //SLM�ł̉���f��(4�Ŋ���鐮���Ɍ���)
//#define PJRSY 500     //SLM�ł̏c��f��(4�Ŋ���鐮���Ɍ���)

#define short 540     //PJRSY��PJRSX�̒Z��
//#define short 500     //PJRSY��PJRSX�̒Z��

#define N 70       //�摜�̖���
#define LENS_SIZE 60 //�g�U�����Y�̃����Y�T�C�Y
//#define LENS_SIZE 25

#define CHECK_NUM N  //�V�~�����[�V�����摜���`�F�b�N����ԍ�
#define lam 532e-09  //�g��
#define d 1.496e-05 //��f�s�b�`
//#define d 6e-05
#define a 0.1 //�`������1
#define b 0.03 //�`������2
#define f 0.03 //�œ_����
//#define a 0.2
//#define b 3
//#define f 0.2
#define resolution pow(2, 8) //�𑜓x
#define approx true    //�����Y�̎��̋ߎ�

//�t�@�C���p�X
string binpath = "../../../../dat/bindat/1byte/fm_28_1.dat";
string simpath = "../../../../dat/simdat/SLM_phase/1byte/lsd/test_sim.dat";
string oriimg = "./test.bmp";
string simimg = "./testsim.bmp";
string t = "exp.bmp";

int main() {
    clock_t start, lap;
    start = clock();

    //�ǂݍ��݃o�C�g�m�F
    int byte_num;
    do {
        cout << "\n�o�C�i���f�[�^��4�o�C�g�œǂݍ��݁F4�����\t1�o�C�g�œǂݍ��݁F1�����\n";
        cout << " 1 or 4: "; cin >> byte_num;

    } while (byte_num != 4 && byte_num != 1);


    //�������݃o�C�g�m�F
    int byte_numw;
    do {
        cout << "\n�o�C�i���f�[�^��4�o�C�g�ŏ������݁F4�����\t1�o�C�g�ŏ������݁F1�����\n";
        cout << " 1 or 4: "; cin >> byte_numw;

    } while (byte_numw != 4 && byte_numw != 1);



    //�摜�f�[�^��U�����(����)�Ƃ��邩�A�ʑ��ɕϊ����邩�m�F
    int ampl_or_phase;
    do {
        cout << "\n�摜�f�[�^��U�����(����)�Ƃ���ꍇ�F0�����\t���K����A�ʑ����Ƃ���ꍇ�F1�����\n";
        cout << " 0 or 1 : "; cin >> ampl_or_phase;

    } while (ampl_or_phase != 0 && ampl_or_phase != 1);



    //�����_���ʑ��������Y�A���C���m�F
    int rand_or_lsd;
    do {
        cout << "\n�g�U�������_���ʑ��Ƃ���ꍇ�F0�����\t�g�U����������Y�A���C�Ƃ���ꍇ�F1�����\n";
        cout << " 0 or 1 : "; cin >> rand_or_lsd;

    } while (rand_or_lsd != 0 && rand_or_lsd != 1);

    //�t�@�C�����́E�o�C�i���X�g���[���I�[�v��
    ifstream ifs(binpath, ios::binary /*| ios::in*/);
    //�t�@�C���o�́E�o�C�i���X�g���[���I�[�v��
    ofstream ofs(simpath, ios::binary/* | ios::out*/);



    //�����I�[�v���ł������m�F
    if ((ifs) && (ofs)) {

        My_LensArray* Lens;
        Lens = new My_LensArray(SX * SY, SX, SY, approx, f, lam, d);

        if (rand_or_lsd == 0) {
            //�����_���g�U��
            Lens->diffuser_Random(0);

        }
        else {
            //�����Y�A���C�g�U��
            Lens->diffuser_Lensarray(LENS_SIZE);


        }


        //H�z�񒼐ڌv�Z
        //��ʑ̂���g�U��
        My_ComArray_2D* Ha, * Hb;
        Ha = new My_ComArray_2D(4 * SX * SY, 2 * SX, 2 * SY);

        Ha->H_kaku(lam, a, d);


        //��ʑ̂���Z���T
        Hb = new My_ComArray_2D(4 * SX * SY, 2 * SX, 2 * SY);

        Hb->H_kaku(lam, b, d);



        for (int k = 0; k < N; k++) {
            //�i���󋵕\��
            if (k == 0) {
                cout << "\n\n\n-------------------------------simdata�t�@�C���쐬��---------------------------------\n\n\n";
            }

            //�o�C�i���ǂݍ��ݔz��|�C���^
            unsigned char* chRe;
            int* intRe;



            chRe = new unsigned char[BX * BY];
            intRe = new int[BX * BY];



            //data�ǂݎ��
            if (byte_num == 1) {
                //1byte�ňꖇ���ǂݍ���
                ifs.read((char*)chRe, sizeof(unsigned char) * BX * BY);
                //�㉺���]
                invert_img<unsigned char>(chRe, chRe, BX, BY);

            }
            else {
                //4byte�ňꖇ���ǂݍ���
                ifs.read((char*)intRe, sizeof(int) * BX * BY);
                //�㉺���]
                invert_img<int>(intRe, intRe, BX, BY);


            }


            //�摜�f�[�^�m�F
            if (k == N - 1) {

                My_Bmp* check;
                check = new My_Bmp(BX, BY);

                if (byte_num == 1) {

                    check->uc_to_img(chRe);
                    check->img_write(oriimg);
                }
                else {
                    check->data_to_ucimg(intRe);
                    check->img_write(oriimg);

                }



                delete check;
            }
            delete[]intRe;

            //�摜�f�[�^���g�傷��Ƃ�CV_8U�ł��
            //�摜�f�[�^��cv::Mat�ɃR�s�[
            Mat bin_mat(BY, BX, CV_8U);
            memcpy(bin_mat.data, chRe, BX * BY * sizeof(unsigned char));
            /*imshow("View", bin_mat);
            waitKey(0);*/
            delete[]chRe;

            //�g��
            Mat bin_mat_res(short, short, CV_8U);
            resize(bin_mat, bin_mat_res, Size(short, short));
            bin_mat.release();
            /*string resizeimg = "resize.bmp";
            imwrite(resizeimg, bin_mat_res);
            imshow("View", bin_mat_res);
            waitKey(0);*/

            //�[�����߂��č��킹��
            Mat bin_mat_pjr(SY, SX, CV_8U);
            copyMakeBorder(bin_mat_res, bin_mat_pjr, (int)(SY - short) / 2, (int)(SY - short) / 2, (int)(SX - short) / 2, (int)(SX - short) / 2, BORDER_CONSTANT, 0);
            bin_mat_res.release();
            /*string padimg = "pad.bmp";
            imwrite(padimg, bin_mat_pjr);
            imshow("View", bin_mat_pjr);
            waitKey(0);*/

            unsigned char* padRe;
            padRe = new unsigned char[SX * SY];


            //�g�債��cv::Mat��padRe�ɃR�s�[
            memcpy(padRe, bin_mat_pjr.data, SX * SY * sizeof(unsigned char));
            bin_mat_pjr.release();



            //�摜�f�[�^�m�F
            if (k == N - 1) {


                My_Bmp* check;
                check = new My_Bmp(SX, SY);

                check->uc_to_img(padRe);
                check->img_write(t);

                delete check;

            }



            My_ComArray_2D* Complex;
            Complex = new My_ComArray_2D(SX * SY, SX, SY);

            Complex->data_to_ReIm(padRe);



            delete[]padRe;


            if (ampl_or_phase == 1) {
                //�ʑ����ɂ���
                Complex->to_phase(Complex->Re);
            }



            //�g�U�܂ł̓`���v�Z
            Ha->kaku(Complex, Complex);

            //�g�U��X�摜
            Complex->mul_complex(Lens);

            //���C���Z���T�܂œ`���v�Z
            Hb->kaku(Complex, Complex);



            //�U���v�Z
            Complex->power(Complex->Re);

            if (k == CHECK_NUM - 1) {

                My_Bmp* check;
                check = new My_Bmp(SX, SY);

                check->data_to_ucimg(Complex->Re);
                check->img_write(simimg);

                delete check;

            }


            double* Pline;
            Pline = new double[SX];

            mid_line<double>(Complex->Re, SX, SY, Pline);
            delete Complex;

            //�������ݔz��
            int* intw;
            unsigned char* chw;
            intw = new int[SX];
            chw = new unsigned char[SX];

            norm_reso_n<double>(Pline, intw, (int)(resolution - 1), SX);
            delete[]Pline;
            to_uch(intw, SX, chw);


            //��������
            if (byte_numw == 1) {
                ofs.write((char*)chw, sizeof(unsigned char) * SX);

            }
            else {
                ofs.write((char*)intw, sizeof(int) * SX);

            }


            delete[]intw;
            delete[]chw;


            if ((k + 1) % 100 == 0) {
                cout << k + 1 << "�܂Ŋ���----------------------------------------------\n";
                lap = clock();
                cout << setprecision(4) << (double)(lap - start) / CLOCKS_PER_SEC / 60 << "���o��\n\n";

            }
        }
        delete Lens;

        delete Ha;
        delete Hb;

    }

    else {
        cout << "�f�[�^�t�@�C�����J���܂���ł���\n�I�����܂��B";

    }

    return 0;
}