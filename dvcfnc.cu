#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>

#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"
#include "dvcfnc.cuh"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

//copy
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

//�������C�u�����C���N���[�h
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
//copy

//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)

using namespace std;
using namespace cv;

//CUDA
#ifndef __HIPCC__
#define __HIPCC__
#endif 

#define sqr(x) ((x)*(x))


//�֐��Q

//void set_cufftcomplex(hipfftComplex* cuconp, double* Re, double* Im, int size) {
//    for (int i = 0; i < size; i++) {
//        cuconp[i] = make_hipComplex((float)Re[i], (float)Im[i]);
//    }
//}

//void set_cufftcomplex(hipComplex* cuconp, double* Re, double* Im, int size) {
//    for (int i = 0; i < size; i++) {
//        cuconp[i] = make_hipComplex((float)Re[i], (float)Im[i]);
//    }
//}


__global__ void cusetcufftcomplex(hipComplex* com, double* Re, double* Im, int size)
{

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < size) {
        com[idx] = make_hipComplex((float)Re[idx], (float)Im[idx]);
    }
}


__global__ void normfft(hipfftComplex* dev, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < x * y) {
        dev[idx] = make_hipComplex(hipCrealf(dev[idx]) / (x * y), hipCimagf(dev[idx]) / (x * y));
    }
}



void fft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
    hipfftHandle plan;

    //hipfftPlan2d ��2���� : �ł��x���ω����鎟���̃T�C�Y
    //hipfftPlan2d ��3���� : �ł������ω����鎟���̃T�C�Y
    hipfftPlan2d(&plan, y, x, HIPFFT_C2C);
    hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}


void ifft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
    hipfftHandle plan;

    //hipfftPlan2d ��2���� : �ł��x���ω����鎟���̃T�C�Y
    //hipfftPlan2d ��3���� : �ł������ω����鎟���̃T�C�Y
    hipfftPlan2d(&plan, y, x, HIPFFT_C2C);
    hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
    hipfftDestroy(plan);
}

void cufftcom2mycom(My_ComArray_2D* out, hipfftComplex* in, int s) {
    for (int i = 0; i < s; i++) {
        out->Re[i] = (double)hipCrealf(in[i]);
        out->Im[i] = (double)hipCimagf(in[i]);

    }
}



__global__ void Hcudaf(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {
        Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
        Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
    }
}


__global__ void HcudacuCom(hipComplex* H, int x, int y, float z, float d, float lam)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    float u = 1 / (x * d), v = 1 / (y * d);


    if (idy < y && idx < x) {
        H[idy * x + idx] = make_hipComplex(cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2)))),
            sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2)))));
    }
}


//void Hnotgpu(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
//{
//    for (int i = 0; i < y; i++) {
//        for (int j = 0; j < x; j++) {
//            Re[i * x + j] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)j - x / 2)) - sqr(v * ((float)i - y / 2))));
//            Im[i * x + j] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)j - x / 2)) - sqr(v * ((float)i - y / 2))));
//        }
//    }
//}


__global__ void  shiftf(float* ore, float* oim, float* re, float* im, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {

        if (idx < x / 2 && idy < y / 2) {
            ore[idy * x + idx] = re[(idy + y / 2) * x + (idx + x / 2)];
            ore[(idy + y / 2) * x + (idx + x / 2)] = re[idy * x + idx];
            oim[idy * x + idx] = im[(idy + y / 2) * x + (idx + x / 2)];
            oim[(idy + y / 2) * x + (idx + x / 2)] = im[idy * x + idx];
        }
        else if (idx >= x / 2 && idy < y / 2) {
            ore[idy * x + idx] = re[(idy + y / 2) * x + (idx - x / 2)];
            ore[(idy + y / 2) * x + (idx - x / 2)] = re[idy * x + idx];
            oim[idy * x + idx] = im[(idy + y / 2) * x + (idx - x / 2)];
            oim[(idy + y / 2) * x + (idx - x / 2)] = im[idy * x + idx];
        }
    }
}

__global__ void shiftCom(hipComplex* out, hipComplex* in, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {

        if (idx < x / 2 && idy < y / 2) {
            out[idy * x + idx] = in[(idy + y / 2) * x + (idx + x / 2)];
            out[(idy + y / 2) * x + (idx + x / 2)] = in[idy * x + idx];

        }
        else if (idx >= x / 2 && idy < y / 2) {
            out[idy * x + idx] = in[(idy + y / 2) * x + (idx - x / 2)];
            out[(idy + y / 2) * x + (idx - x / 2)] = in[idy * x + idx];

        }
    }
}


//void shiftnotgpu(float* ore, float* oim, float* re, float* im, int x, int y) {
//
//    for (int i = 0; i < y; i++) {
//        for (int j = 0; j < x; j++) {
//            if (j < x / 2 && i < y / 2) {
//                ore[i * x + j] = re[(i + y / 2) * x + (j + x / 2)];
//                ore[(i + y / 2) * x + (j + x / 2)] = re[i * x + j];
//                oim[i * x + j] = im[(i + y / 2) * x + (j + x / 2)];
//                oim[(i + y / 2) * x + (j + x / 2)] = im[i * x + j];
//            }
//            else if (j >= x / 2 && i < y / 2) {
//                ore[i * x + j] = re[(i + y / 2) * x + (j - x / 2)];
//                ore[(i + y / 2) * x + (j - x / 2)] = re[i * x + j];
//                oim[i * x + j] = im[(i + y / 2) * x + (j - x / 2)];
//                oim[(i + y / 2) * x + (j - x / 2)] = im[i * x + j];
//            }
//        }
//    }
//}

//floatXcufftCom
__global__ void mulcomcufftcom(hipfftComplex* out, float* re, float* im, hipfftComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex(re[idx] * hipCrealf(in[idx]) - im[idx] * hipCimagf(in[idx]),
            re[idx] * hipCimagf(in[idx]) + im[idx] * hipCrealf(in[idx]));

    }
}


//doubleXcufftCom
__global__ void muldoublecomcufftcom(hipfftComplex* out, double* re, double* im, hipfftComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex((float)re[idx] * hipCrealf(in[idx]) - (float)im[idx] * hipCimagf(in[idx]),
            (float)re[idx] * hipCimagf(in[idx]) + (float)im[idx] * hipCrealf(in[idx]));

    }
}

__global__ void Cmulfft(hipfftComplex* out, hipfftComplex* fin, hipComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    hipfftComplex tmp1, tmp2;
    tmp1 = make_hipComplex(hipCrealf(fin[idx]), hipCimagf(fin[idx]));
    tmp2 = make_hipComplex(hipCrealf(in[idx]), hipCimagf(in[idx]));

    if (idx < s) {

        out[idx] = hipCmulf(tmp1, tmp2);

    }

}




__global__ void pad_cufftcom2cufftcom(hipfftComplex* out, int lx, int ly, hipfftComplex* in, int sx, int sy)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < sx && idy < sy) {
        out[(idy + ly / 4) * lx + (idx + lx / 4)] = in[idy * sx + idx];
    }

}


__global__ void elimpad(hipfftComplex* out, int sx, int sy, hipfftComplex* in, int lx, int ly)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < sx && idy < sy) {
        out[idy * sx + idx] = in[(idy + ly / 4) * lx + (idx + lx / 4)];
    }
}

//
//
//void kaku_cuda(hipfftComplex* devicein, float* ReHs, float* ImHs, int x, int y, dim3 grid, dim3 block, int mulgrid, int mulblock) {
//
//    hipfftComplex* devpad;
//    hipMalloc((void**)&devpad, sizeof(hipfftComplex) * 4 * x * y);
//    hipMemset(devpad, 0, sizeof(hipfftComplex) * 4 * x * y);
//
//    pad_cufftcom2cufftcom << <grid, block >> > (devpad, 2 * x, 2 * y, devicein, x, y);
//
//
//
//    fft_2D_cuda_dev(2 * x, 2 * y, devpad);
//
//
//    //�|���Z
//    hipfftComplex* rslt;
//    hipMalloc((void**)&rslt, sizeof(hipfftComplex) * x * y * 4);
//    mulcomcufftcom << <mulgrid, mulblock >> > (rslt, ReHs, ImHs, devpad, 4 * x * y);
//
//
//    ifft_2D_cuda_dev(2 * x, 2 * y, rslt);
//
//
//
//    //devicein��0elim
//    elimpad << <grid, block >> > (devicein, x, y, rslt, 2 * x, 2 * y);
//
//
//
//    hipFree(devpad);
//    hipFree(rslt);
//
//}
//
//void kakucuda(hipfftComplex* devin, int inx, int iny, hipfftComplex* devpad, hipfftComplex* mul, float* ReHs, float* ImHs
//    , dim3 gridpadelim, dim3 block, int mulgrid, int mulblock) {
//
//    hipMemset(devpad, 0, sizeof(hipfftComplex) * 4 * inx * iny);
//    pad_cufftcom2cufftcom<<<gridpadelim, block >>>(devpad, 2 * inx, 2 * iny, devin, inx, iny);
//    fft_2D_cuda_dev(2 * inx, 2 * iny, devpad);
//    mulcomcufftcom<<<mulgrid, mulblock >> > (mul, ReHs, ImHs, devpad, 4 * inx * iny);
//    ifft_2D_cuda_dev(2 * inx, 2 * iny, mul);
//    elimpad<<<gridpadelim, block >>>(devin, inx, iny, mul, 2 * inx, 2 * iny);
//}
//

void Hcudaf_shiftf(float* devReH, float* devImH, int x, int y, float d, float z, float lamda, dim3 grid, dim3 block) {
    float* ReH, * ImH;
    hipMalloc((void**)&ReH, sizeof(float) * x * y);
    hipMalloc((void**)&ImH, sizeof(float) * x * y);

    float u = 1 / (x * d), v = 1 / (y * d);

    Hcudaf << <grid, block >> > (ReH, ImH, x, y, u, v, z, lamda);
    shiftf << <grid, block >> > (devReH, devImH, ReH, ImH, x, y);

    hipFree(ReH);
    hipFree(ImH);
}

void Hcudashiftcom(hipComplex* dev, int x, int y, float z, float d, float lamda, dim3 grid, dim3 block) {
    hipComplex* tmp;
    hipMalloc((void**)&tmp, sizeof(hipComplex) * x * y);

    HcudacuCom << <grid, block >> > (tmp, x, y, z, d, lamda);
    shiftCom << <grid, block >> > (dev, tmp, x, y);

    hipFree(tmp);

}


__global__ void cucompower(double* power, hipComplex* dev, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        power[idx] = sqrt((double)sqr(hipCrealf(dev[idx])) + (double)sqr(hipCimagf(dev[idx])));

    }
}





__global__ void cunormaliphase(hipComplex* out, double* normali, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex((float)cos(2 * M_PI * normali[idx]), (float)sin(2 * M_PI * normali[idx]));

    }

}



//void Hnotgpushift(float* devReH, float* devImH, int x, int y, float d, float z, float lamda, dim3 grid, dim3 block) {
//   /* float* ReH, * ImH;
//    hipMalloc((void**)&ReH, sizeof(float) * x * y);
//    hipMalloc((void**)&ImH, sizeof(float) * x * y);*/
//
//    float* Re, * Im, * Res, * Ims;
//    Re = new float[x * y];
//    Im = new float[x * y];
//    Res = new float[x * y];
//    Ims = new float[x * y];
//
//    float u = 1 / (x * d), v = 1 / (y * d);
//
//    Hnotgpu(Re, Im, x, y, u, v, z, lamda);
//
//    shiftnotgpu(Res, Ims, Re, Im, x, y);
//
//    //hipMemcpy(ReH, Re, sizeof(float) * x * y, hipMemcpyHostToDevice);
//    //hipMemcpy(ImH, Im, sizeof(float) * x * y, hipMemcpyHostToDevice);
//
//    //shiftf << <grid, block >> > (devReH, devImH, ReH, ImH, x, y);
//
//    /*hipFree(ReH);
//    hipFree(ImH);*/
//
//    hipMemcpy(devReH, Res, sizeof(float) * x * y, hipMemcpyHostToDevice);
//    hipMemcpy(devImH, Ims, sizeof(float) * x * y, hipMemcpyHostToDevice);
//
//
//    delete[]Re;
//    delete[]Im;
//    delete[]Res;
//    delete[]Ims;
//
//}

