#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include "my_all.h"
#include "Bmp_class_dll.h"
#include "complex_array_class_dll.h"

#include <opencv2//opencv.hpp>
#include <iostream>
#include <fstream>
#include <string>

#ifndef __HIPCC__
#define __HIPCC__
#endif 

#define sqr(x) ((x)*(x))
#define SX 512
#define SY 512


//�ǉ��̈ˑ��t�@�C���ݒ�̑���
//opencv��DLL��PATH��ʂ��ē��I�����N���C�u����(�ÖٓI�����N)�Ƃ���
#pragma comment(lib, "opencv_world454.lib")
#pragma comment(lib, "opencv_world454d.lib")


//bmp�N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "Dll_bmp_class.lib")
//���f�z��N���X�𓮓I�����N(�ÖٓI�����N)
#pragma comment(lib, "DllComArray.lib")

//�]���Ȍx���폜
#pragma warning(disable:4996)

using namespace std;
using namespace cv;

string impath = "./lena512x512.bmp";
string ompath = "./output.bmp";


void set_cufftcomplex(hipfftComplex* cuconp, double* Re, double* Im, int size) {
	for (int i = 0; i < size; i++) {
		cuconp[i] = make_hipComplex((float)Re[i], (float)Im[i]);
	}
}

void set_Re_cufftcomplex(hipfftComplex* cuconp, double* Re, int size) {
	for (int i = 0; i < size; i++) {
		cuconp[i] = make_hipComplex((float)Re[i], 0.0f);
	}
}

//CUDA_FFT2D
void fft_2D_cuda(int x, int y, hipfftComplex* host) {
	hipfftHandle plan;

	hipfftComplex* dev;

	hipMalloc((void**)&dev, sizeof(hipfftComplex) * x * y);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * x * y, hipMemcpyHostToDevice);

	//�t�[���G�ϊ�
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);

	hipMemcpy(host, dev, sizeof(hipfftComplex) * x * y, hipMemcpyDeviceToHost);


	hipFree(dev);
	hipfftDestroy(plan);
}

//CUDA_IFFT2D
void ifft_2D_cuda(int x, int y, hipfftComplex* host) {
	hipfftHandle plan;

	hipfftComplex* dev;

	hipMalloc((void**)&dev, sizeof(hipfftComplex) * x * y);
	hipMemcpy(dev, host, sizeof(hipfftComplex) * x * y, hipMemcpyHostToDevice);

	//�t�[���G�ϊ�
	hipfftPlan2d(&plan, x, y, HIPFFT_C2C);
	hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);

	hipMemcpy(host, dev, sizeof(hipfftComplex) * x * y, hipMemcpyDeviceToHost);

	hipFree(dev);
	hipfftDestroy(plan);
}

__global__ void pad(double* out, int x, int y, double* in)
{
	int X = 2 * x;
	int Y = 2 * y;
	
	//���͂��ꂽ�摜�f�[�^���O���߂��Ĕ{�̑傫���̉摜�ɂ���
	for (int i = Y / 4; i < y + Y / 4; i++) {
		for (int j = X / 4; j < x + X / 4; j++) {
			out[i * X + j] = in[(i - Y / 4) * x + (j - X / 4)];
		}
	}
}




void kaku(double* dev2, double* dev)
{
	dim3 grid(1, 1);
	dim3 block(1, 1);
	pad << <grid, block >> > (dev2, SX, SY, dev);
}


int main(void) {
	/*cv::Mat inimg = cv::imread(impath);
	cv::imshow("lena", inimg);


	memcpy(padRe, bin_mat_pjr.data, PJRSX * PJRSY * sizeof(unsigned char));
	bin_mat_pjr.release();




	cv::Mat outimg;
	cvtColor(inimg, outimg, COLOR_BGR2GRAY);


	cv::waitKey(0);
	cv::imwrite(ompath, outimg);*/


    //���͉摜��ǂ݂���
	My_Bmp* inimg;
	inimg = new My_Bmp(SX, SY);

	inimg->img_read(impath);

	////�摜���畡�f�z��Ɉړ�
	//My_ComArray_2D* com;
	//com = new My_ComArray_2D(SX * SY, SX, SY);

	//inimg->ucimg_to_double(com->Re);

	////�z�X�g���������m��
	//hipfftComplex* host;
	//host = (hipfftComplex*)malloc(sizeof(hipfftComplex) * SX * SY);
	////���f�z��̎������Z�b�g
	//set_Re_cufftcomplex(host, com->Re, SX * SY);

	double* host;
	host = (double*)malloc(sizeof(double) * SX * SY);

	inimg->ucimg_to_double(host);


	double* dev;
	hipMalloc((void**)&dev, sizeof(double) * SX * SY);
	hipMemcpy(dev, host, sizeof(double) * SX * SY, hipMemcpyHostToDevice);


	//cout << host[0];

	double* host2;
	host2 = (double*)malloc(sizeof(double) * SX * SY * 4);
	memset(host2, 0, sizeof(double) * SX * SY * 4);

	double* dev2;
	hipMalloc((void**)&dev2, sizeof(double) * SX * SY *4);
	hipMemcpy(dev2, host2, sizeof(double) * SX * SY * 4, hipMemcpyHostToDevice);

	
	kaku(dev2, dev);


	////FFT
	//fft_2D_cuda(SX, SY, host);

	////IFFT
	//ifft_2D_cuda(SX, SY, host);


	
	
	/*for (int i = 0; i < SX * SY; i++) {
		com->Re[i] = (double)sqrt(sqr(hipCrealf(host[i])) + sqr(hipCimagf(host[i])));
	}*/


	hipMemcpy(host2, dev2, sizeof(double) * 4 * SX * SY, hipMemcpyDeviceToHost);

	
	//cout << host2[500000];


	My_Bmp* outimg;
	outimg = new My_Bmp(SX * 2, SY * 2);

	outimg->data_to_ucimg(host2);
	outimg->img_write(ompath);

	delete inimg;
	//delete com;
	delete outimg;
	free(host);
	return 0;
}